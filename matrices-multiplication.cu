#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define CHECK_CUDA_CALL(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)


/**
* Multiplies two matrices A and B and stores result in matrix C.
* 
* @param A matrix A
* @param B matrix B
* @param C resulting matrix C
* @param n size of square matrices
*/
void 
multiplyMatricesCPU(
    float* A, 
    float* B, 
    float* C, 
    int n
) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            C[i * n + j] = 0;
            for (int k = 0; k < n; ++k) {
                C[i * n + j] += A[i * n + k] * B[k * n + j];
            }
        }
    }
}


__global__ 
void 
multiplyMatricesGPU(
    float* A, 
    float* B, 
    float* C, 
    int n
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}


void 
transposeMatrix(
    float* B, 
    float* B_T, 
    int n
) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            B_T[j * n + i] = B[i * n + j];
        }
    }
}


void 
multiplyMatricesWithTransposeCPU(
    float* A, 
    float* B_T, 
    float* C, 
    int n
) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            C[i * n + j] = 0;
            for (int k = 0; k < n; ++k) {
                C[i * n + j] += A[i * n + k] * B_T[j * n + k];
            }
        }
    }
}

void 
printMatrix(float* matrix, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << matrix[i * n + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

int 
main() {
    int sizes[] = {3, 512, 102, 2048 };

    for (int i = 0; i < 4; ++i) {
        int N = sizes[i];

        float* A = new float[N * N];
        float* B = new float[N * N];
        float* C_CPU = new float[N * N];
        float* C_GPU = new float[N * N];
        float* C_Trans = new float[N * N];

        for (int i = 0; i < N * N; ++i) {
            A[i] = static_cast<float>(rand()) / RAND_MAX;
            B[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        if (N == 3) {
            printMatrix(A, N);
            printMatrix(B, N);
        }

        // CPU Time Measurement
        auto startCPU = std::chrono::high_resolution_clock::now();
        multiplyMatricesCPU(A, B, C_CPU, N);
        auto endCPU = std::chrono::high_resolution_clock::now();
        double timeCPU = std::chrono::duration<double, std::milli>(endCPU - startCPU).count();
        
        if (N == 3) {
            std::cout << "Calculated on CPU:\n";
            printMatrix(C_CPU, N);
        }

        float* d_A, * d_B, * d_C;
        CHECK_CUDA_CALL(hipMalloc(&d_A, N * N * sizeof(float)));
        CHECK_CUDA_CALL(hipMalloc(&d_B, N * N * sizeof(float)));
        CHECK_CUDA_CALL(hipMalloc(&d_C, N * N * sizeof(float)));
        CHECK_CUDA_CALL(hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_CALL(hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice));

        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

        // GPU Time Measurement with CUDA Events
        hipEvent_t startGPU, endGPU;
        CHECK_CUDA_CALL(hipEventCreate(&startGPU));
        CHECK_CUDA_CALL(hipEventCreate(&endGPU));

        CHECK_CUDA_CALL(hipEventRecord(startGPU));
        multiplyMatricesGPU << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
        CHECK_CUDA_CALL(hipDeviceSynchronize());
        CHECK_CUDA_CALL(hipEventRecord(endGPU));

        CHECK_CUDA_CALL(hipEventSynchronize(endGPU));

        float elapsedGPU;
        CHECK_CUDA_CALL(hipEventElapsedTime(&elapsedGPU, startGPU, endGPU));
        double timeGPU = elapsedGPU;

        CHECK_CUDA_CALL(hipMemcpy(C_GPU, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));

        if (N == 3) {
            std::cout << "Calculated on GPU:\n";
            printMatrix(C_GPU, N);
        }

        float* B_T = new float[N * N];
        transposeMatrix(B, B_T, N);
        auto startTrans = std::chrono::high_resolution_clock::now();
        multiplyMatricesWithTransposeCPU(A, B_T, C_Trans, N);
        auto endTrans = std::chrono::high_resolution_clock::now();
        double timeTrans = std::chrono::duration<double, std::milli>(endTrans - startTrans).count();

        if (N == 3) {
            std::cout << "Calculated on CPU with transpose:\n";
            printMatrix(C_Trans, N);
        }

        std::cout << "Matrix size: " << N << "x" << N << "\n";
        std::cout << "Time (CPU): " << timeCPU << " ms\n";
        std::cout << "Time (GPU): " << timeGPU << " ms\n";
        std::cout << "Time (CPU with Transpose): " << timeTrans << " ms\n";

        double slowest = std::max({ timeCPU, timeGPU, timeTrans });
        std::cout << "Speedup (CPU vs GPU): " << slowest / timeGPU << "x\n";
        std::cout << "Speedup (CPU vs Transpose): " << slowest / timeTrans << "x\n\n";

        delete[] A;
        delete[] B;
        delete[] C_CPU;
        delete[] C_GPU;
        delete[] C_Trans;
        delete[] B_T;
        CHECK_CUDA_CALL(hipFree(d_A));
        CHECK_CUDA_CALL(hipFree(d_B));
        CHECK_CUDA_CALL(hipFree(d_C));

        CHECK_CUDA_CALL(hipEventDestroy(startGPU));
        CHECK_CUDA_CALL(hipEventDestroy(endGPU));
    }
    return 0;
}
